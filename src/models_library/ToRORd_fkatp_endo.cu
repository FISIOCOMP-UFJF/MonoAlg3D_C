#include "hip/hip_runtime.h"
#include "ToRORd_fkatp_endo.h"
#include <stddef.h>
#include <stdint.h>

#define DT *((real *)((char *)sv + pitch * 43) + thread_id)
#define TIME_NEW *((real *)((char *)sv + pitch * 44) + thread_id)
#define PREVIOUS_DT *((real *)((char *)sv + pitch * 45) + thread_id)

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    uint8_t use_adpt_h = (uint8_t)solver->adaptive;

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(use_adpt), &use_adpt_h, sizeof(uint8_t)));
    log_to_stdout_and_file("Using ToRORd_fkatp_endo GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_h) {
        real reltol_h = solver->rel_tol;
        real abstol_h = solver->abs_tol;
        real max_dt_h = solver->max_dt;
        real min_dt_h = solver->min_dt;

        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(reltol), &reltol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(abstol), &abstol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(max_dt), &max_dt_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(min_dt), &min_dt_h, sizeof(real)));
        log_to_stdout_and_file("Using Adaptive Euler model to solve the ODEs\n");
    } else {
        log_to_stdout_and_file("Using Euler model to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_h)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes);

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();
    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES_GPU(solve_model_odes_gpu) {

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(
            hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve,
                                    num_steps);

    check_cuda_error(hipPeekAtLastError());

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device)
        check_cuda_error(hipFree(cells_to_solve_device));
}

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadID < num_volumes) {

        *((real *)((char *)sv + pitch * 0) + threadID) = -8.876380e+01f; // v millivolt
        *((real *)((char *)sv + pitch * 1) + threadID) = 1.110000e-02f;  // CaMKt millimolar
        *((real *)((char *)sv + pitch * 2) + threadID) = 1.210250e+01f;  // nai millimolar
        *((real *)((char *)sv + pitch * 3) + threadID) = 1.210290e+01f;  // nass millimolar
        *((real *)((char *)sv + pitch * 4) + threadID) = 1.423002e+02f;  // ki millimolar
        *((real *)((char *)sv + pitch * 5) + threadID) = 1.423002e+02f;  // kss millimolar
        *((real *)((char *)sv + pitch * 6) + threadID) = 8.158300e-05f;  // cai millimolar
        *((real *)((char *)sv + pitch * 7) + threadID) = 7.030500e-05f;  // cass millimolar
        *((real *)((char *)sv + pitch * 8) + threadID) = 1.521100e+00f;  // cansr millimolar
        *((real *)((char *)sv + pitch * 9) + threadID) = 1.521400e+00f;  // cajsr millimolar
        *((real *)((char *)sv + pitch * 10) + threadID) = 8.057200e-04f; // m dimensionless
        *((real *)((char *)sv + pitch * 11) + threadID) = 8.286000e-01f; // h dimensionless
        *((real *)((char *)sv + pitch * 12) + threadID) = 8.284000e-01f; // j dimensionless
        *((real *)((char *)sv + pitch * 13) + threadID) = 6.707000e-01f; // hp dimensionless
        *((real *)((char *)sv + pitch * 14) + threadID) = 8.281000e-01f; // jp dimensionless
        *((real *)((char *)sv + pitch * 15) + threadID) = 1.629000e-04f; // mL dimensionless
        *((real *)((char *)sv + pitch * 16) + threadID) = 5.255000e-01f; // hL dimensionless
        *((real *)((char *)sv + pitch * 17) + threadID) = 2.872000e-01f; // hLp dimensionless
        *((real *)((char *)sv + pitch * 18) + threadID) = 9.509800e-04f; // a dimensionless
        *((real *)((char *)sv + pitch * 19) + threadID) = 9.996000e-01f; // iF dimensionless
        *((real *)((char *)sv + pitch * 20) + threadID) = 5.936000e-01f; // iS dimensionless
        *((real *)((char *)sv + pitch * 21) + threadID) = 4.845400e-04f; // ap dimensionless
        *((real *)((char *)sv + pitch * 22) + threadID) = 9.996000e-01f; // iFp dimensionless
        *((real *)((char *)sv + pitch * 23) + threadID) = 6.538000e-01f; // iSp dimensionless
        *((real *)((char *)sv + pitch * 24) + threadID) = 8.108400e-09f; // d dimensionless
        *((real *)((char *)sv + pitch * 25) + threadID) = 1.000000e+00f; // ff dimensionless
        *((real *)((char *)sv + pitch * 26) + threadID) = 9.390000e-01f; // fs dimensionless
        *((real *)((char *)sv + pitch * 27) + threadID) = 1.000000e+00f; // fcaf dimensionless
        *((real *)((char *)sv + pitch * 28) + threadID) = 9.999000e-01f; // fcas dimensionless
        *((real *)((char *)sv + pitch * 29) + threadID) = 1.000000e+00f; // jca dimensionless
        *((real *)((char *)sv + pitch * 30) + threadID) = 1.000000e+00f; // ffp dimensionless
        *((real *)((char *)sv + pitch * 31) + threadID) = 1.000000e+00f; // fcafp dimensionless
        *((real *)((char *)sv + pitch * 32) + threadID) = 6.646200e-04f; // nca_ss dimensionless
        *((real *)((char *)sv + pitch * 33) + threadID) = 1.200000e-03f; // nca_i dimensionless
        *((real *)((char *)sv + pitch * 34) + threadID) = 9.981000e-01f; // C3 dimensionless
        *((real *)((char *)sv + pitch * 35) + threadID) = 8.510900e-04f; // C2 dimensionless
        *((real *)((char *)sv + pitch * 36) + threadID) = 7.034400e-04f; // C1 dimensionless
        *((real *)((char *)sv + pitch * 37) + threadID) = 3.758500e-04f; // O dimensionless
        *((real *)((char *)sv + pitch * 38) + threadID) = 1.328900e-05f; // I dimensionless
        *((real *)((char *)sv + pitch * 39) + threadID) = 2.480000e-01f; // xs1 dimensionless
        *((real *)((char *)sv + pitch * 40) + threadID) = 1.770700e-04f; // xs2 dimensionless
        *((real *)((char *)sv + pitch * 41) + threadID) = 1.612900e-22f; // Jrel_np millimolar_per_millisecond
        *((real *)((char *)sv + pitch * 42) + threadID) = 1.247500e-20f; // Jrel_p millimolar_per_millisecond

        if(use_adpt) {
            *((real *)((char *)sv + pitch * 43) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 44) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 45) + threadID) = 0.0;    // previous dt
        }
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve,
                          uint32_t num_cells_to_solve, int num_steps) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real *)((char *)sv + pitch * i) + sv_id) =
                        dt * rDY[i] + *((real *)((char *)sv + pitch * i) + sv_id);
                }
            }
        } else {
            solve_Forward_Euler_gpu_adpt(sv, stim_currents[threadID], cur_time + max_dt, sv_id);
        }
    }
}

inline __device__ void solve_Forward_Euler_gpu_adpt(real *sv, real stim_curr, real final_time, int thread_id) {

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = powf(abstol, 2.0f);

    // dt = ((time_new + dt) > final_time) ? (final_time - time_new) : dt;
    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
    time_new += dt;

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

    int count = 0;

    int count_limit = (final_time - time_new) / min_dt;

    int aux_count_limit = count_limit + 2000000;

    if(aux_count_limit > 0) {
        count_limit = aux_count_limit;
    }

    while(1) {

        for(int i = 0; i < NEQ; i++) {
            // stores the old variables in a vector
            edos_old_aux_[i] = sv_local[i];
            // //computes euler method
            edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
            // steps ahead to compute the rk2 method
            sv_local[i] = edos_new_euler_[i];
        }

        time_new += dt;

        RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
        time_new -= dt; // step back

        real greatestError = 0.0, auxError = 0.0;
        //#pragma unroll
        for(int i = 0; i < NEQ; i++) {

            // stores the new evaluation
            _k2__[i] = rDY[i];
            _aux_tol = fabs(edos_new_euler_[i]) * reltol;
            _tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

            // finds the greatest error between  the steps
            auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

            greatestError = (auxError > greatestError) ? auxError : greatestError;
        }

        /// adapt the time step
        greatestError += __tiny_;
        previous_dt = dt;
        /// adapt the time step
        dt = _beta_safety_ * dt * sqrtf(1.0f / greatestError);

        if(time_new + dt > final_time) {
            dt = final_time - time_new;
        }

        // it doesn't accept the solution
        if(count < count_limit && (greatestError >= 1.0f)) {
            // restore the old values to do it again
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_old_aux_[i];
            }
            count++;
            // throw the results away and compute again
        } else {
            count = 0;

            // if(greatestError >=1.0) {
            //    printf("Thread //d,accepting solution with error > //lf \n", threadID, greatestError);
            //}

            // it accepts the solutions
            // int aux = (dt > max_step && max_step != 0);
            // dt = (aux) ? max_step : dt;

            if(dt < min_dt) {
                dt = min_dt;
            }

            else if(dt > max_dt && max_dt != 0) {
                dt = max_dt;
            }

            if(time_new + dt > final_time) {
                dt = final_time - time_new;
            }

            // change vectors k1 e k2 , para que k2 seja aproveitado como k1 na proxima iteração
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                _k_aux__[i] = _k2__[i];
                _k2__[i] = _k1__[i];
                _k1__[i] = _k_aux__[i];
            }

            // it steps the method ahead, with euler solution
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_new_euler_[i];
            }

            // verifica se o incremento para a próxima iteração ultrapassa o tempo de salvar, q neste caso é o tempo
            // final
            if(time_new + previous_dt >= final_time) {
                // se são iguais, ja foi calculada a iteração no ultimo passo de tempo e deve-se para o laço
                // nao usar igualdade - usar esta conta, pode-se mudar a tolerância
                // printf("//d: //lf\n", threadID, fabs(final_time - time_new));
                if((fabs(final_time - time_new) < 1.0e-5)) {
                    break;
                } else if(time_new < final_time) {
                    dt = previous_dt = final_time - time_new;
                    time_new += previous_dt;
                    break;
                } else {
                    dt = previous_dt = min_dt;
                    time_new += (final_time - time_new);
                    printf("Nao era pra chegar aqui: %d: %lf\n", thread_id, final_time - time_new);
                    break;
                }
            } else {
                time_new += previous_dt;
            }
        }
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, int threadID_, real dt) {

    // State variables
    real v_old_;
    real CaMKt_old_;
    real nai_old_;
    real nass_old_;
    real ki_old_;
    real kss_old_;
    real cai_old_;
    real cass_old_;
    real cansr_old_;
    real cajsr_old_;
    real m_old_;
    real h_old_;
    real j_old_;
    real hp_old_;
    real jp_old_;
    real mL_old_;
    real hL_old_;
    real hLp_old_;
    real a_old_;
    real iF_old_;
    real iS_old_;
    real ap_old_;
    real iFp_old_;
    real iSp_old_;
    real d_old_;
    real ff_old_;
    real fs_old_;
    real fcaf_old_;
    real fcas_old_;
    real jca_old_;
    real ffp_old_;
    real fcafp_old_;
    real nca_ss_old_;
    real nca_i_old_;
    real C3_old_;
    real C2_old_;
    real C1_old_;
    real O_old_;
    real I_old_;
    real xs1_old_;
    real xs2_old_;
    real Jrel_np_old_;
    real Jrel_p_old_;

    if(use_adpt) {
        v_old_ = sv[0];
        CaMKt_old_ = sv[1];
        nai_old_ = sv[2];
        nass_old_ = sv[3];
        ki_old_ = sv[4];
        kss_old_ = sv[5];
        cai_old_ = sv[6];
        cass_old_ = sv[7];
        cansr_old_ = sv[8];
        cajsr_old_ = sv[9];
        m_old_ = sv[10];
        h_old_ = sv[11];
        j_old_ = sv[12];
        hp_old_ = sv[13];
        jp_old_ = sv[14];
        mL_old_ = sv[15];
        hL_old_ = sv[16];
        hLp_old_ = sv[17];
        a_old_ = sv[18];
        iF_old_ = sv[19];
        iS_old_ = sv[20];
        ap_old_ = sv[21];
        iFp_old_ = sv[22];
        iSp_old_ = sv[23];
        d_old_ = sv[24];
        ff_old_ = sv[25];
        fs_old_ = sv[26];
        fcaf_old_ = sv[27];
        fcas_old_ = sv[28];
        jca_old_ = sv[29];
        ffp_old_ = sv[30];
        fcafp_old_ = sv[31];
        nca_ss_old_ = sv[32];
        nca_i_old_ = sv[33];
        C3_old_ = sv[34];
        C2_old_ = sv[35];
        C1_old_ = sv[36];
        O_old_ = sv[37];
        I_old_ = sv[38];
        xs1_old_ = sv[39];
        xs2_old_ = sv[40];
        Jrel_np_old_ = sv[41];
        Jrel_p_old_ = sv[42];
    } else {
        //    //State variables
        v_old_ = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt_old_ = *((real *)((char *)sv + pitch * 1) + threadID_);
        nai_old_ = *((real *)((char *)sv + pitch * 2) + threadID_);
        nass_old_ = *((real *)((char *)sv + pitch * 3) + threadID_);
        ki_old_ = *((real *)((char *)sv + pitch * 4) + threadID_);
        kss_old_ = *((real *)((char *)sv + pitch * 5) + threadID_);
        cai_old_ = *((real *)((char *)sv + pitch * 6) + threadID_);
        cass_old_ = *((real *)((char *)sv + pitch * 7) + threadID_);
        cansr_old_ = *((real *)((char *)sv + pitch * 8) + threadID_);
        cajsr_old_ = *((real *)((char *)sv + pitch * 9) + threadID_);
        m_old_ = *((real *)((char *)sv + pitch * 10) + threadID_);
        h_old_ = *((real *)((char *)sv + pitch * 11) + threadID_);
        j_old_ = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp_old_ = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp_old_ = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL_old_ = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL_old_ = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp_old_ = *((real *)((char *)sv + pitch * 17) + threadID_);
        a_old_ = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF_old_ = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS_old_ = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap_old_ = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp_old_ = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp_old_ = *((real *)((char *)sv + pitch * 23) + threadID_);
        d_old_ = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff_old_ = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs_old_ = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf_old_ = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas_old_ = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca_old_ = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp_old_ = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp_old_ = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca_ss_old_ = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i_old_ = *((real *)((char *)sv + pitch * 33) + threadID_);
        C3_old_ = *((real *)((char *)sv + pitch * 34) + threadID_);
        C2_old_ = *((real *)((char *)sv + pitch * 35) + threadID_);
        C1_old_ = *((real *)((char *)sv + pitch * 36) + threadID_);
        O_old_ = *((real *)((char *)sv + pitch * 37) + threadID_);
        I_old_ = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1_old_ = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2_old_ = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np_old_ = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p_old_ = *((real *)((char *)sv + pitch * 42) + threadID_);
    }

    #include "ToROrd_common.inc.c"
}