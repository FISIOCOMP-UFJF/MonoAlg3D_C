#include "hip/hip_runtime.h"
#include "Maleckar2008.h"
#include <stddef.h>
#include <stdint.h>

__constant__  size_t pitch;
__constant__  real abstol;
__constant__  real reltol;
__constant__  real max_dt;
__constant__  real min_dt;
__constant__  uint8_t use_adpt;
size_t pitch_h;

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    uint8_t use_adpt_h = (uint8_t)solver->adaptive;

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(use_adpt), &use_adpt_h, sizeof(uint8_t)));
    log_to_stdout_and_file("Using Maleckar2008 GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_h) {
        real reltol_h = solver->rel_tol;
        real abstol_h = solver->abs_tol;
        real max_dt_h = solver->max_dt;
        real min_dt_h = solver->min_dt;

        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(reltol), &reltol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(abstol), &abstol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(max_dt), &max_dt_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(min_dt), &min_dt_h, sizeof(real)));
        log_to_stdout_and_file("Using Adaptive Euler model to solve the ODEs\n");
    } else {
        log_to_stdout_and_file("Using Euler model to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_h)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes);

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();
    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(
            hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve,
                                    num_steps);

    check_cuda_error(hipPeekAtLastError());

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device)
        check_cuda_error(hipFree(cells_to_solve_device));
}

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
         *((real * )((char *) sv + pitch * 0) + threadID)  = -87.169816169406;
         *((real * )((char *) sv + pitch * 1) + threadID)  = 0.001075453357;
         *((real * )((char *) sv + pitch * 2) + threadID)  = 0.990691306716;
         *((real * )((char *) sv + pitch * 3) + threadID)  = 0.993888937283;
         *((real * )((char *) sv + pitch * 4) + threadID)  = 0.000018211252;
         *((real * )((char *) sv + pitch * 5) + threadID)  = 0.979322592773;
         *((real * )((char *) sv + pitch * 6) + threadID)  = 0.001208153482;
         *((real * )((char *) sv + pitch * 7) + threadID)  = 0.000033616596;
         *((real * )((char *) sv + pitch * 8) + threadID)  = 0.004173008466;
         *((real * )((char *) sv + pitch * 9) + threadID)  = 0.015242594688;
         *((real * )((char *) sv + pitch * 10) + threadID) = 0.007074239331;
         *((real * )((char *) sv + pitch * 11) + threadID) = 0.048267587131;
         *((real * )((char *) sv + pitch * 12) + threadID) = 0.105468807033;
         *((real * )((char *) sv + pitch * 13) + threadID) = 0.00364776906;
         *((real * )((char *) sv + pitch * 14) + threadID) = 0.174403618112;
         *((real * )((char *) sv + pitch * 15) + threadID) = 0.003643592594;
         *((real * )((char *) sv + pitch * 16) + threadID) = 0.993331326442;
         *((real * )((char *) sv + pitch * 17) + threadID) = 97.505463697266;
         *((real * )((char *) sv + pitch * 18) + threadID) = 0.006679257264;
         *((real * )((char *) sv + pitch * 19) + threadID) = 11.441712311614;
         *((real * )((char *) sv + pitch * 20) + threadID) = 1.716573130685;
         *((real * )((char *) sv + pitch * 21) + threadID) = 0.226941113355;
         *((real * )((char *) sv + pitch * 22) + threadID) = 0.256752008084;
         *((real * )((char *) sv + pitch * 23) + threadID) = 104.450004990523;
         *((real * )((char *) sv + pitch * 24) + threadID) = 22.171689894953;
         *((real * )((char *) sv + pitch * 25) + threadID) = 19.864701949854;

        if(use_adpt) {
            *((real *)((char *)sv + pitch * 26) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 27) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 28) + threadID) = 0.0;    // previous dt
        }
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve,
                          uint32_t num_cells_to_solve, int num_steps) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real *)((char *)sv + pitch * i) + sv_id) =
                        dt * rDY[i] + *((real *)((char *)sv + pitch * i) + sv_id);
                }
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], cur_time + max_dt, sv_id);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real final_time, int thread_id) {

    #define DT *((real *)((char *)sv + pitch * 30) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * 31) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * 32) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = powf(abstol, 2.0f);

    // dt = ((time_new + dt) > final_time) ? (final_time - time_new) : dt;
    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
    time_new += dt;

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

    int count = 0;

    int count_limit = (final_time - time_new) / min_dt;

    int aux_count_limit = count_limit + 2000000;

    if(aux_count_limit > 0) {
        count_limit = aux_count_limit;
    }

    while(1) {

        for(int i = 0; i < NEQ; i++) {
            // stores the old variables in a vector
            edos_old_aux_[i] = sv_local[i];
            // //computes euler method
            edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
            // steps ahead to compute the rk2 method
            sv_local[i] = edos_new_euler_[i];
        }

        time_new += dt;

        RHS_gpu(sv_local, rDY, stim_curr, thread_id, dt);
        time_new -= dt; // step back

        real greatestError = 0.0, auxError = 0.0;
        for(int i = 0; i < NEQ; i++) {

            // stores the new evaluation
            _k2__[i] = rDY[i];
            _aux_tol = fabs(edos_new_euler_[i]) * reltol;
            _tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

            // finds the greatest error between  the steps
            auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

            greatestError = (auxError > greatestError) ? auxError : greatestError;
        }

        /// adapt the time step
        greatestError += __tiny_;
        previous_dt = dt;
        /// adapt the time step
        dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

        if(time_new + dt > final_time) {
            dt = final_time - time_new;
        }

        // it doesn't accept the solution
        if(count < count_limit && (greatestError >= 1.0f)) {
            // restore the old values to do it again
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_old_aux_[i];
            }
            count++;
            // throw the results away and compute again
        } else {
            count = 0;

            if(dt < min_dt) {
                dt = min_dt;
            }

            else if(dt > max_dt && max_dt != 0) {
                dt = max_dt;
            }

            if(time_new + dt > final_time) {
                dt = final_time - time_new;
            }

            // change vectors k1 e k2 , para que k2 seja aproveitado como k1 na proxima iteração
            for(int i = 0; i < NEQ; i++) {
                _k_aux__[i] = _k2__[i];
                _k2__[i] = _k1__[i];
                _k1__[i] = _k_aux__[i];
            }

            // it steps the method ahead, with euler solution
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_new_euler_[i];
            }

            if(time_new + previous_dt >= final_time) {
                if((fabs(final_time - time_new) < 1.0e-5)) {
                    break;
                } else if(time_new < final_time) {
                    dt = previous_dt = final_time - time_new;
                    time_new += previous_dt;
                    break;
                } else {
                    dt = previous_dt = min_dt;
                    time_new += (final_time - time_new);
                    printf("Error: %d: %lf\n", thread_id, final_time - time_new);
                    break;
                }
            } else {
                time_new += previous_dt;
            }
        }
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}



inline __device__ void RHS_gpu(real *sv, real *rDY, real stim_current, int thread_id, real dt) {
    //State variables
    double var_cell__V ;  // Units: mV; Initial value: -87.169816169406
    double var_INa__xm ; // Units: dimensionless; Initial value: 0.001075453357
    double var_INa__xh ;  // Units: dimensionless; Initial value: 0.990691306716
    double var_INa__xj ; // Units: dimensionless; Initial value: 0.993888937283
    double var_ICaL__c1 ; // Units: dimensionless; Initial value: 0.000018211252
    double var_ICaL__c2 ; // Units: dimensionless; Initial value: 0.979322592773
    double var_ICaL__xi1ca ; // Units: dimensionless; Initial value: 0.001208153482
    double var_ICaL__xi1ba ; // Units: dimensionless; Initial value: 0.000033616596
    double var_ICaL__xi2ca ; // Units: dimensionless; Initial value: 0.004173008466
    double var_ICaL__xi2ba ; // Units: dimensionless; Initial value: 0.015242594688
    double var_IKr__xr ; // Units: dimensionless; Initial value: 0.007074239331
    double var_IKs__xs1 ; // Units: dimensionless; Initial value: 0.048267587131
    double var_IKs__xs2 ;  // Units: dimensionless; Initial value: 0.105468807033
    double var_Ito__xtos ;  // Units: dimensionless; Initial value: 0.00364776906
    double var_Ito__ytos ; // Units: dimensionless; Initial value: 0.174403618112
    double var_Ito__xtof ; // Units: dimensionless; Initial value: 0.003643592594
    double var_Ito__ytof ; // Units: dimensionless; Initial value: 0.993331326442
    double var_Irel__Ca_JSR ; // Units: uM; Initial value: 97.505463697266
    double var_Irel__xir ; // Units: uM_per_ms; Initial value: 0.006679257264
    double var_Na__Na_i ; // Units: mM; Initial value: 11.441712311614
    double var_Ca__Ca_dyad ; // Units: uM; Initial value: 1.716573130685
    double var_Ca__Ca_submem ; // Units: uM; Initial value: 0.226941113355
    double var_Ca__Ca_i ; // Units: uM; Initial value: 0.256752008084
    double var_Ca__Ca_NSR ; // Units: uM; Initial value: 104.450004990523
    double var_Ca__tropi ;// Units: uM; Initial value: 22.171689894953
    double var_Ca__trops ; // Units: uM; Initial value: 19.864701949854
    
    if(use_adpt) {
        var_cell__V = sv[0];
        var_INa__xm = sv[1];
        var_INa__xh = sv[2];
        var_INa__xj = sv[3];
        var_ICaL__c1 = sv[4];
        var_ICaL__c2 = sv[5];
        var_ICaL__xi1ca = sv[6];
        var_ICaL__xi1ba = sv[7];
        var_ICaL__xi2ca = sv[8];
        var_ICaL__xi2ba = sv[9];
        var_IKr__xr = sv[10];
        var_IKs__xs1 = sv[11];
        var_IKs__xs2 = sv[12];
        var_Ito__xtos = sv[13];
        var_Ito__ytos = sv[14];
        var_Ito__xtof = sv[15];
        var_Ito__ytof = sv[16];
        var_Irel__Ca_JSR = sv[17];
        var_Irel__xir = sv[18];
        var_Na__Na_i = sv[19];
        var_Ca__Ca_dyad = sv[20];
        var_Ca__Ca_submem = sv[21];
        var_Ca__Ca_i = sv[22];
        var_Ca__Ca_NSR = sv[23];
        var_Ca__tropi = sv[24];
        var_Ca__trops = sv[25];
    } else {
        var_cell__V =  *((real*)((char*)sv + pitch * 0) + thread_id);
        var_INa__xm =  *((real*)((char*)sv + pitch * 1) + thread_id);
        var_INa__xh =  *((real*)((char*)sv + pitch * 2) + thread_id);
        var_INa__xj =  *((real*)((char*)sv + pitch * 3) + thread_id);
        var_ICaL__c1 =  *((real*)((char*)sv + pitch * 4) + thread_id);
        var_ICaL__c2 =  *((real*)((char*)sv + pitch * 5) + thread_id);
        var_ICaL__xi1ca =  *((real*)((char*)sv + pitch * 6) + thread_id);
        var_ICaL__xi1ba =  *((real*)((char*)sv + pitch * 7) + thread_id);
        var_ICaL__xi2ca =  *((real*)((char*)sv + pitch * 8) + thread_id);
        var_ICaL__xi2ba =  *((real*)((char*)sv + pitch * 9) + thread_id);
        var_IKr__xr =  *((real*)((char*)sv + pitch * 10) + thread_id);
        var_IKs__xs1 =  *((real*)((char*)sv + pitch * 11) + thread_id);
        var_IKs__xs2 =  *((real*)((char*)sv + pitch * 12) + thread_id);
        var_Ito__xtos =  *((real*)((char*)sv + pitch * 13) + thread_id);
        var_Ito__ytos =  *((real*)((char*)sv + pitch * 14) + thread_id);
        var_Ito__xtof =  *((real*)((char*)sv + pitch * 15) + thread_id);
        var_Ito__ytof =  *((real*)((char*)sv + pitch * 16) + thread_id);
        var_Irel__Ca_JSR =  *((real*)((char*)sv + pitch * 17) + thread_id);
        var_Irel__xir =  *((real*)((char*)sv + pitch * 18) + thread_id);
        var_Na__Na_i =  *((real*)((char*)sv + pitch * 19) + thread_id);
        var_Ca__Ca_dyad =  *((real*)((char*)sv + pitch * 20) + thread_id);
        var_Ca__Ca_submem =  *((real*)((char*)sv + pitch * 21) + thread_id);
        var_Ca__Ca_i =  *((real*)((char*)sv + pitch * 22) + thread_id);
        var_Ca__Ca_NSR =  *((real*)((char*)sv + pitch * 23) + thread_id);
        var_Ca__tropi =  *((real*)((char*)sv + pitch * 24) + thread_id);
        var_Ca__trops =  *((real*)((char*)sv + pitch * 25) + thread_id);
    }

    #include "Maleckar2008_common.inc.c"

}

